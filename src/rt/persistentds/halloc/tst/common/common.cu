#include "hip/hip_runtime.h"
/** @file common.cu implementation of common library for Halloc testing */

#define COMMONTEST_COMPILING

#include <limits.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/sort.h>

#include "common.h"

using namespace thrust;

// parsing options
void print_option(const char *option, const char *message) {
	printf("  %-14s %s\n", option, message);
}

void print_suboption(const char *suboption, const char *message) {
	printf("      %-10s %s\n", suboption, message);
}

void print_usage_and_exit(int exit_code) {
	//printf("%s", opts_usage_g);
	printf("usage: <test-name> [<optitons>]\noptions:\n");
	print_option("-h", "print this message and exit");
	print_option("-a <allocator>", "select the allocator:");
	print_suboption("cuda", "CUDA allocator");
	print_suboption("halloc", "Halloc allocator (default)");
#ifdef WITH_SCATTER
	print_suboption("scatter", "ScatterAlloc allocator");
#endif
	print_option("-m <nbytes>", "heap size, default 512MiB");
	print_option("-C <fraction>", "heap fraction reserved for halloc, " 
							 "default 0.75");
	print_option("-B <fraction>", "busy block occupancy, default 0.835");
	print_option("-R <fraction>", "roomy block occupancy, default 0.6");
	print_option("-b <power>", "log2 of slab size, default 22 (=4MiB)");
	print_option("-D <device>", "GPU to run on, default 0");
	print_option("-n <nthreads>", "kernel grid size in threads, default 1M");
	print_option("-t <ntries>", "number of test tries, default 8");
	print_option("-T <nthreads>", "thread block size, default 128");
	print_option("-s <nbytes>", "minimum allocation size, default 16");
	print_option("-S <nbytes>", "maximum allocation size, default 16");
	printf("if only one of -s and -S is set to some value, the other " 
				 "one is also set to that value\n");
	// TODO: remarks when only -s or -S is set
	print_option("-l <nallocs>", "number of successive mallocs, default 4");
	print_option("-i <niters>", "number of iterations inside kernel, default 1");
	print_option("-q <power>", "log2 of allocation thread period, default 0");
	print_option("-g <power>", "log2 of RNG thread period, default 0");
	print_option("-d <distr>", "allocation size distribution:");
	print_suboption("uniform", "uniform size distribution (default)");
	print_suboption("expuniform", "f*2^e, f in [0,1], f and e uniform");
	print_suboption("expequal", "f*2^e, f uniform in [0,1], e geometric");
	print_option("-f <fraction>", "end of state 0 allocated fraction, default 1");
	print_option("-F <fraction>", "end of state 1 allocated fraction, default 0");
	print_option("-e <fraction>", "fraction of threads doing someting, " 
							 "default 1");
	//printf("other options to be added later\n");
	exit(exit_code);
}  // print_usage_and_exit

double parse_double(char *str, double a = 0.0, double b = 1.0) {
	double r;
	if(sscanf(str, "%lf", &r) != 1) {
		fprintf(stderr, "%s is not a double value\n", str);
		print_usage_and_exit(-1);
	}
	if(r < a || r > b) {
		fprintf(stderr, "double value %lf is not in range [%lf, %lf]\n", r, a, b);
		print_usage_and_exit(-1);
	}
	return r;
}  // parse_double

int parse_int(char *str, int a = INT_MIN, int b = INT_MAX) {
	int r;
	if(sscanf(str, "%d", &r) != 1) {
		fprintf(stderr, "%s is not an integer value or too big\n", &r);
		print_usage_and_exit(-1);
	}
	if(r < a || r > b) {
		fprintf(stderr, "integer value %d is not in range [%d, %d]\n", r, a, b);
		print_usage_and_exit(-1);
	}
	return r;
}  // parse_int

char *allocator_types[] = {
	"cuda", "halloc", "scatter", "xmalloc"
};

char *distr_types[] = {
	"uniform", "expuniform", "expequal"
};

static uint parse_enum(char *str, char *name, char **vals, uint top) {
	int istr;
	for(istr = 0; istr < top - 1; istr++)
		if(!strcmp(str, vals[istr]))
			break;
	istr++;
	if(istr == top) {
		printf("%s: invalid %s name\n", str, name);
		print_usage_and_exit(-1);
	}
	return istr;
}  // parse_enum

AllocatorType parse_allocator(char *str) {
	return (AllocatorType)parse_enum
		(str, "allocator", allocator_types, AllocatorTopNone);
}  // parse_allocator

DistrType parse_distr(char *str) {
	return (DistrType)parse_enum
		(str, "distribution", distr_types, DistrTopNone);
}  // parse_distr

void CommonOpts::parse_cmdline(int argc, char **argv) {
	static const char *common_opts_str = ":ha:m:C:B:R:D:b:n:t:T:s:S:l:i:q:g:d:f:F:e:";
	int c;
	int period_sh, ndevices;
	cucheck(hipGetDeviceCount(&ndevices));
	bool nthreads_explicit = false, min_alloc_explicit = false, 
		max_alloc_explicit = false;
	while((c = getopt(argc, argv, common_opts_str)) != -1) {
		switch(c) {
			// general options (and errors)
		case 'h':
			print_usage_and_exit(0);
			break;
		case ':':
			fprintf(stderr, "missing argument for option %c\n", optopt);
			print_usage_and_exit(-1);
			break;
		case '?':
			fprintf(stderr, "unknown option -%c\n", optopt);
			print_usage_and_exit(-1);
			break;

			// allocator options
		case 'a':
			allocator = parse_allocator(optarg);
			break;
		case 'm':
			memory = parse_int(optarg, 4096);
			break;
		case 'C':
			halloc_fraction = parse_double(optarg);
			break;
		case 'B':
			busy_fraction = parse_double(optarg);
			break;
		case 'R':
			roomy_fraction = parse_double(optarg);
			break;
		case 'b':
			sb_sz_sh = parse_int(optarg, 20, 26);
			break;

			// test options
		case 'D':
			device = parse_int(optarg, 0, ndevices - 1);
			break;
		case 'n':
			nthreads = parse_int(optarg, 0);
			nthreads_explicit = true;
			break;
		case 't':
			ntries = parse_int(optarg, 1);
			break;
		case 'T':
			bs = parse_int(optarg, 1, 1024);
			break;
		case 's':
			min_alloc_explicit = true;
			alloc_sz = parse_int(optarg, 0);
			if(max_alloc_explicit) { 
				if(max_alloc_sz < alloc_sz) {
					fprintf(stderr, "max allocation size should be >= " 
									"min allocation	size\n");
					print_usage_and_exit(-1);
				}
			} else
				max_alloc_sz = alloc_sz;
			break;
		case 'S':
			max_alloc_explicit = true;
			//printf("before setting max_alloc_sz = %d\n", max_alloc_sz);
			max_alloc_sz = parse_int(optarg, 0);
			//printf("after setting max_alloc_sz = %d\n", max_alloc_sz);
			if(min_alloc_explicit) {
				if(max_alloc_sz < alloc_sz) {
					fprintf(stderr, "max allocation size should be >= " 
									"min allocation	size\n");
					print_usage_and_exit(-1);
				}
			} else
				alloc_sz = max_alloc_sz;
			break;
		case 'l':
			nallocs = parse_int(optarg, 1);
			break;
		case 'i':
			niters = parse_int(optarg, 1);
			break;
		case 'q':
			period_sh = parse_int(optarg, 0, 31);
			period_mask = period_sh > 0 ? ((1 << period_sh) - 1) : 0;
			break;
		case 'g':
			group_sh = parse_int(optarg, 0, 31);
			break;
		case 'd':
			distr_type = parse_distr(optarg);
			break;
		case 'f':
			alloc_fraction = (float)parse_double(optarg);
			break;
		case 'F':
			free_fraction = (float)parse_double(optarg);
			break;
		case 'e':
			exec_fraction = (float)parse_double(optarg);
			break;
			
		default:
			fprintf(stderr, "this simply should not happen when parsing options\n");
			print_usage_and_exit(-1);
			break;
		}  // switch
	}

	// cap memory to fraction of device memory
	int device;
	cucheck(hipGetDevice(&device));
	hipDeviceProp_t props;
	cucheck(hipGetDeviceProperties(&props, device));
	size_t dev_memory = props.totalGlobalMem;
	memory = min((unsigned long long)memory, 
							 (unsigned long long)(0.75 * dev_memory));

	// cap number of threads for CUDA allocator
	if(allocator == AllocatorCuda && !nthreads_explicit)
		nthreads = min(nthreads, 32 * 1024);
	// check probabilities
	// if(palloc + pfree > 1) {
	// 	printf("palloc = %lf, pfree = %lf, total > 1\n", (double)palloc, 
	// 				 (double)pfree);
	// 	print_usage_and_exit(-1);
	// }

	// recompute some fields
	recompute_fields();
	//printf("min_sz = %d, max_sz = %d\n", alloc_sz, max_alloc_sz);
}  // parse_cmdline

double CommonOpts::expected_sz(void) {
	if(alloc_sz == max_alloc_sz)
		return alloc_sz;
	switch(distr_type) {
	case DistrUniform:
		return ((double)alloc_sz + max_alloc_sz) / 2;
	case DistrExpUniform:
		{
			double expectation = 0;
			for(uint sh = 0; sh <= max_alloc_sh; sh++) {
				double lo = alloc_sz << sh;
				double hi = min((alloc_sz << (sh + 1)) - 1, max_alloc_sz);
				expectation += (lo + hi) / 2;
			}
			expectation /= max_alloc_sh + 1;
			return expectation;
		}
	case DistrExpEqual:
		{
			double expectation = 0, probab = 1;
			for(uint sh = 0; sh <= max_alloc_sh; sh++) {
				if(sh < max_alloc_sz)
					probab /= 2;
				double lo = alloc_sz << sh;
				double hi = min((alloc_sz << (sh + 1)) - 1, max_alloc_sz);
				expectation += (lo + hi) / 2 * probab;
			}
			//expectation /= max_alloc_sh + 1;
			return expectation;
		}
	default:
		// this shouldn't happen
		fprintf(stderr, "invalid distribution type\n");
		exit(-1);
	}  // switch
}

double CommonOpts::total_nallocs(void) {
	return (double)nptrs_cont(nthreads) * nallocs * niters * ntries;
}

double CommonOpts::total_sz(void) {
	return expected_sz() * total_nallocs();
}

void CommonOpts::recompute_fields(void) {
	// recompute max_alloc_sh
	max_alloc_sh = 0;
	while(max_alloc_sz >= alloc_sz << (max_alloc_sh + 1))
		max_alloc_sh++;

	// recompute probabilities
	if(exec_fraction + alloc_fraction + free_fraction > 2) {
		fprintf(stderr, "too large change fraction\n");
		print_usage_and_exit(-1);
	} else if(exec_fraction < fabsf(alloc_fraction - free_fraction)) {
		fprintf(stderr, "too small change fraction\n");
		print_usage_and_exit(-1);
	}
	// 0 = alloc, 1 = free
	float p00, p01, p10, p11;
	if(free_fraction < 1) {
		p00 = (exec_fraction + alloc_fraction - free_fraction) / 
			(2 * (1 - free_fraction));
	} else
		p00 = 1;
	if(free_fraction > 0) {
		p01 = (exec_fraction - alloc_fraction + free_fraction) / 
		  (2 * free_fraction);
	} else
		p01 = 1;
	if(alloc_fraction < 1) {
		p10 = (exec_fraction - alloc_fraction + free_fraction) / 
			(2 * (1 - alloc_fraction));
	} else
		p10 = 1;
	if(alloc_fraction > 0) {
		p11 = (exec_fraction + alloc_fraction - free_fraction) / 
			(2 * alloc_fraction);
	}
	probabs[0][0] = p00;
	probabs[0][1] = p01;
	probabs[1][0] = p10;
	probabs[1][1] = p11;
	//printf("p00 = %.2lf, p01 = %.2lf, p02 = %.2lf, p03 = %.2lf\n",
	//			 (double)p00, (double)p01, (double)p10, (double)p11);
}  // recompute_fields

void drandom_init(const CommonOpts &opts) {
	srandom((uint)time(0));
	//srandom(12345);

	// TODO: somehow standardize this number
	const uint MAX_NTHREADS = 8 * 1024 * 1024;
	uint n = max(MAX_NTHREADS, opts.nthreads);
	size_t sz = n * sizeof(uint);
	uint *d_random_states, *h_random_states;

	// allocate memory
	cucheck(hipMalloc((void **)&d_random_states, sz));
	h_random_states = (uint *)malloc(sz);

	// initialize random values, respect groups
	uint gp = opts.group() * opts.period();
	uint seed;
	for(uint i = 0; i < n; i++) {
		if(i % gp == 0)
			seed = random();
		h_random_states[i] = seed;
	}
	cucheck(hipMemcpy(d_random_states, h_random_states, sz, 
										 hipMemcpyHostToDevice));
	free(h_random_states);
	
	// initialize device variable
	cuset(random_states_g, uint *, d_random_states);	
}  // drandom_init

void drandom_shutdown(const CommonOpts &opts) {
	// currently nothing is done
}

struct ptr_is_nz {
	void **ptrs;
	uint *ctrs;
	CommonOpts opts;
	__host__ __device__ ptr_is_nz
	(void **ptrs, uint *ctrs, const CommonOpts &opts) 
		: opts(opts), ptrs(ptrs), ctrs(ctrs) {}
	__host__ __device__ bool operator()(int i) { 
		if(opts.is_thread_inactive(i)) 
			return true;
		else {
			uint ctr = ctrs ? ctrs[i] : 1;
			for(uint ialloc = 0; ialloc < ctr; ialloc++) {
				if(!ptrs[ialloc * opts.nthreads + i])
					return false;
			}
			return true;
		}
	}  // operator ()
};  // ptr_is_nz

bool check_nz(void **d_ptrs, uint *d_ctrs, uint nptrs, const CommonOpts &opts) {
	return all_of
		(counting_iterator<int>(0), counting_iterator<int>(nptrs),
		 ptr_is_nz(d_ptrs, d_ctrs, opts));
}  // check_nz

__global__ void copy_cont_k
(void **to, void **from, uint *ctrs, uint *fill_ctr, CommonOpts opts) {
	uint i = threadIdx.x + blockIdx.x * blockDim.x;
	if(opts.is_thread_inactive(i))
		return;
	uint nallocs = ctrs ? ctrs[i] : opts.nallocs;
	uint pos = atomicAdd(fill_ctr, nallocs);
	for(uint ialloc = 0; ialloc < nallocs; ialloc++)
		to[pos + ialloc] = from[ialloc * opts.nthreads + i];
}  // copy_cont_k

/** a helper functor to check whether each pointer has enough room */
struct has_enough_room {
	uint64 *d_ptrs;
	uint alloc_sz;
	int nptrs;
	__host__ __device__ has_enough_room
	(uint64 *d_ptrs, uint alloc_sz, int	nptrs) 
		: d_ptrs(d_ptrs), alloc_sz(alloc_sz), nptrs(nptrs) {}
	__host__ __device__ bool operator()(int i) {
		if(i == nptrs - 1)
			return true;
		uint sz = alloc_sz ? alloc_sz : *(uint *)(void *)d_ptrs[i];
		bool res = d_ptrs[i] + sz <= d_ptrs[i + 1];
		// if(!res) {
		// 	printf("ptrs[%d] = %llx, sz = %d, ptrs[%d] = %llx\n", i, d_ptrs[i], sz, 
		// 				 i + 1, d_ptrs[i + 1]);
		// }
		return res;
	}
};  // has_enough_room

/** a kernel which simply writes thread id at the address specified by each
		pointer in the passed array */
__global__ void write_tid_k(void **d_ptrs, int nptrs) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= nptrs)
		return;
	*(int *)d_ptrs[tid] = tid;
}  // write_tid_k

/** a helper functor to check tid written at each address */
struct check_tid {
	void **d_ptrs;
	__host__ __device__ check_tid(void **d_ptrs) : d_ptrs(d_ptrs) {}
	__host__ __device__ bool operator()(int tid) {
		return *(int *)d_ptrs[tid] == tid;
	}
};

bool check_alloc
(void **d_ptrs, uint *d_ctrs, uint nptrs, const CommonOpts &opts) {
	uint alloc_sz = opts.alloc_sz;
	if(opts.alloc_sz != opts.max_alloc_sz)
		alloc_sz = 0;
	//uint period = opts.period();
	if(!check_nz(d_ptrs, d_ctrs, nptrs, opts)) {
		fprintf(stderr, "cannot allocate enough memory\n");
		return false;
	}
	// first copy into a contiguous location
	uint group = opts.group();
	int nptrs_cont = 0;
	if(d_ctrs) {
		device_ptr<uint> dt_ctrs(d_ctrs);
		nptrs_cont = reduce(dt_ctrs, dt_ctrs + opts.nthreads, 0, plus<uint>());
		//printf("contiguous number of pointers = %d\n", nptrs_cont);
	} else {
		nptrs_cont = opts.nptrs_cont(nptrs / opts.nallocs) * opts.nallocs;
	}
	// check if there are any pointers at all
	if(nptrs_cont == 0)
		return true;
	void **d_ptrs_cont = 0;
	cucheck(hipMalloc((void **)&d_ptrs_cont, nptrs_cont * sizeof(void *)));

	uint *d_fill_ctr;
	cucheck(hipMalloc((void **)&d_fill_ctr, sizeof(uint)));
	cucheck(hipMemset(d_fill_ctr, 0, sizeof(uint)));
	uint bs = 128;
	copy_cont_k<<<divup(opts.nthreads, bs), bs>>>
		(d_ptrs_cont, d_ptrs, d_ctrs, d_fill_ctr, opts);
	cucheck(hipGetLastError());
	cucheck(hipStreamSynchronize(0));
	
	// transform
	// 	(counting_iterator<int>(0), counting_iterator<int>(nptrs_cont),
	// 	 device_ptr<void *>(d_ptrs_cont), copy_cont(d_ptrs_cont, d_ptrs, opts));
	// sort the pointers
	device_ptr<uint64> dt_ptrs((uint64 *)d_ptrs_cont);
	sort(dt_ptrs, dt_ptrs + nptrs_cont);
	// check whether each pointer has enough room
	if(!all_of(counting_iterator<int>(0), counting_iterator<int>(nptrs_cont),
						 has_enough_room((uint64 *)d_ptrs_cont, alloc_sz, nptrs_cont))) {
		fprintf(stderr, "allocated pointers do not have enough room\n");
		cucheck(hipFree(d_ptrs_cont));
		return false;
	} 

	// writes and reads have already been performed with the pointers, so just
	// ignore this part
	// do write-read test to ensure there are no segfaults
	// write_tid_k<<<divup(nptrs_cont, bs), bs>>>(d_ptrs_cont, nptrs_cont);
	// cucheck(hipGetLastError());
	// cucheck(hipStreamSynchronize(0));
	// bool res = all_of(counting_iterator<int>(0), counting_iterator<int>(nptrs_cont), 
	// 							check_tid(d_ptrs_cont));
	bool res = true;
	cucheck(hipFree(d_ptrs_cont));
	cucheck(hipFree(d_fill_ctr));
	return res;
}  // check_alloc
