#include "hip/hip_runtime.h"
/** @file throughput.cu throughput test for various memory allocators */

#include <common.h>

#include <limits.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/** measures malloc throughput */

template<class T> class ThroughputTest {
	
public:
	void operator()(CommonOpts opts, bool warmup) {
		opts.niters = 1;
		// allocate memory
		if(warmup) {
			opts.nthreads = min(4 * opts.bs, opts.nthreads);
			opts.ntries = 1;
		}
		if(!warmup)
			printf("throughput test\n");
		int n = opts.nthreads, bs = opts.bs, grid = divup(n, bs);
		int nptrs = n * opts.nallocs;
		size_t ptrs_sz = nptrs * sizeof(void *);
		void **d_ptrs;
		cucheck(hipMalloc((void **)&d_ptrs, ptrs_sz));
		cucheck(hipMemset(d_ptrs, 0, ptrs_sz));

		double t_malloc = 0, t_free = 0;

		// do testing
		for(int itry = 0; itry < opts.ntries; itry++) {
			// allocate
			double t_malloc_start = omp_get_wtime();
			malloc_k<T> <<<grid, bs>>>(opts, d_ptrs);
			cucheck(hipGetLastError());
			cucheck(hipStreamSynchronize(0));
			double t_malloc_end = omp_get_wtime();
			t_malloc += t_malloc_end - t_malloc_start;
			// check that pointers are correct
			if(!check_nz(d_ptrs, 0, nptrs, opts)) {
				fprintf(stderr, "cannot allocate enough memory\n");
				exit(-1);
			}
			// free
			double t_free_start = omp_get_wtime();
			free_k<T> <<<grid, bs>>>(opts, d_ptrs);
			cucheck(hipGetLastError());
			cucheck(hipStreamSynchronize(0));
			double t_free_end = omp_get_wtime();
			t_free += t_free_end - t_free_start;
		}  // for(itry)

		// output latency infos
		if(!warmup) {
			double malloc_throughput = opts.total_nallocs() / t_malloc * 1e-6;
			double free_throughput = opts.total_nallocs() / t_free * 1e-6;
			double pair_throughput = opts.total_nallocs() / (t_malloc + t_free) 
				* 1e-6;
			double malloc_speed = opts.total_sz() / t_malloc / NBYTES_IN_GIB;
			double pair_speed = opts.total_sz() / (t_malloc + t_free) / NBYTES_IN_GIB;
			printf("malloc throughput %.2lf Mmallocs/s\n", malloc_throughput);
			printf("free throughput %.2lf Mfrees/s\n", free_throughput);
			printf("pair throughput %.2lf Mpairs/s\n", pair_throughput);
			printf("malloc speed %.2lf GiB/s\n", malloc_speed);
			printf("pair speed %.2lf GiB/s\n", pair_speed);
		}  // output latency infos

		// free memory
		cucheck(hipFree(d_ptrs));		
	}  // operator()
 
};  // LatencyTest

int main(int argc, char **argv) {
	CommonOpts opts(true);
	run_test<ThroughputTest>(argc, argv, opts);
	return 0;
}  // main
