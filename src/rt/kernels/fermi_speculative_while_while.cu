#include "hip/hip_runtime.h"
/*
 *  Copyright (c) 2009-2011, NVIDIA Corporation
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are met:
 *      * Redistributions of source code must retain the above copyright
 *        notice, this list of conditions and the following disclaimer.
 *      * Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimer in the
 *        documentation and/or other materials provided with the distribution.
 *      * Neither the name of NVIDIA Corporation nor the
 *        names of its contributors may be used to endorse or promote products
 *        derived from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 *  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *  DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 *  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 *  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 *  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    GF100-optimized variant of the "Speculative while-while"
    kernel used in:

    "Understanding the Efficiency of Ray Traversal on GPUs",
    Timo Aila and Samuli Laine,
    Proc. High-Performance Graphics 2009
*/

#include "CudaTracerKernels.hpp"

//------------------------------------------------------------------------

#define STACK_SIZE  64  // Size of the traversal stack in local memory.

//------------------------------------------------------------------------

extern "C" __global__ void queryConfig(void)
{
    g_config.bvhLayout = BVHLayout_Compact;
    g_config.blockWidth = 32; // One warp per row.
    g_config.blockHeight = 4; // 4*32 = 128 threads, optimal for GTX480
}

//------------------------------------------------------------------------

TRACE_FUNC
{
    // Traversal stack in CUDA thread-local memory.

    int traversalStack[STACK_SIZE];

    // Live state during traversal, stored in registers.

    int     rayidx;                 // Ray index.
    float   origx, origy, origz;    // Ray origin.
    float   dirx, diry, dirz;       // Ray direction.
    float   tmin;                   // t-value from which the ray starts. Usually 0.
    float   idirx, idiry, idirz;    // 1 / dir
    float   oodx, oody, oodz;       // orig / dir

    char*   stackPtr;               // Current position in traversal stack.
    int     leafAddr;               // First postponed leaf, non-negative if none.
    int     nodeAddr;               // Non-negative: current internal node, negative: second postponed leaf.
    int     hitIndex;               // Triangle index of the closest intersection, -1 if none.
    float   hitT;                   // t-value of the closest intersection.

    // Initialize.
    {
        // Pick ray index.

        rayidx = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y));
        if (rayidx >= numRays)
            return;

        // Fetch ray.

        float4 o = rays[rayidx * 2 + 0];
        float4 d = rays[rayidx * 2 + 1];
        origx = o.x, origy = o.y, origz = o.z;
        dirx = d.x, diry = d.y, dirz = d.z;
        tmin = o.w;

        float ooeps = exp2f(-80.0f); // Avoid div by zero.
        idirx = 1.0f / (fabsf(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
        idiry = 1.0f / (fabsf(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
        idirz = 1.0f / (fabsf(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
        oodx = origx * idirx, oody = origy * idiry, oodz = origz * idirz;

        // Setup traversal.

        traversalStack[0] = EntrypointSentinel; // Bottom-most entry.
        stackPtr = (char*)&traversalStack[0];
        leafAddr = 0;   // No postponed leaf.
        nodeAddr = 0;   // Start from the root.
        hitIndex = -1;  // No triangle intersected so far.
        hitT     = d.w; // tmax
    }

    // Traversal loop.

    while (nodeAddr != EntrypointSentinel)
    {
        // Traverse internal nodes until all SIMD lanes have found a leaf.

        bool searchingLeaf = true;
        while (nodeAddr >= 0 && nodeAddr != EntrypointSentinel)
        {
            // Fetch AABBs of the two child nodes.

            float4* ptr = (float4*)((char*)nodesA + nodeAddr);
            float4 n0xy = ptr[0]; // (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)
            float4 n1xy = ptr[1]; // (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)
            float4 nz   = ptr[2]; // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)

            // Intersect the ray against the child nodes.

            float c0lox = n0xy.x * idirx - oodx;
            float c0hix = n0xy.y * idirx - oodx;
            float c0loy = n0xy.z * idiry - oody;
            float c0hiy = n0xy.w * idiry - oody;
            float c0loz = nz.x   * idirz - oodz;
            float c0hiz = nz.y   * idirz - oodz;
            float c1loz = nz.z   * idirz - oodz;
            float c1hiz = nz.w   * idirz - oodz;
			float c0min = spanBeginFermi(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin);
			float c0max = spanEndFermi  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT);
            float c1lox = n1xy.x * idirx - oodx;
            float c1hix = n1xy.y * idirx - oodx;
            float c1loy = n1xy.z * idiry - oody;
            float c1hiy = n1xy.w * idiry - oody;
			float c1min = spanBeginFermi(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, tmin);
			float c1max = spanEndFermi  (c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, hitT);

            bool traverseChild0 = (c0max >= c0min);
            bool traverseChild1 = (c1max >= c1min);

            // Neither child was intersected => pop stack.

            if (!traverseChild0 && !traverseChild1)
            {
                nodeAddr = *(int*)stackPtr;
                stackPtr -= 4;
            }

            // Otherwise => fetch child pointers.

            else
            {
                int2 cnodes = *(int2*)&ptr[3];
                nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y;

                // Both children were intersected => push the farther one.

                if (traverseChild0 && traverseChild1)
                {
                    if (c1min < c0min)
                        swap(nodeAddr, cnodes.y);
                    stackPtr += 4;
                    *(int*)stackPtr = cnodes.y;
                }
            }

            // First leaf => postpone and continue traversal.

            if (nodeAddr < 0 && leafAddr >= 0)
            {
                searchingLeaf = false;
                leafAddr = nodeAddr;
                nodeAddr = *(int*)stackPtr;
                stackPtr -= 4;
            }

            // All SIMD lanes have found a leaf => process them.

            if (!__any(searchingLeaf))
                break;
        }

        // Process postponed leaf nodes.

        while (leafAddr < 0)
        {
            // Intersect the ray against each triangle using Sven Woop's algorithm.

            for (int triAddr = ~leafAddr;; triAddr += 3)
            {
                // Read first 16 bytes of the triangle.
                // End marker (negative zero) => all triangles processed.

                float4 v00 = tex1Dfetch(t_trisA, triAddr + 0);
                if (__float_as_int(v00.x) == 0x80000000)
                    break;

                // Compute and check intersection t-value.

                float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;
                float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);
                float t = Oz * invDz;

                if (t > tmin && t < hitT)
                {
                    // Compute and check barycentric u.

                    float4 v11 = tex1Dfetch(t_trisA, triAddr + 1);
                    float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;
                    float Dx = dirx*v11.x + diry*v11.y + dirz*v11.z;
                    float u = Ox + t*Dx;

                    if (u >= 0.0f && u <= 1.0f)
                    {
                        // Compute and check barycentric v.

                        float4 v22 = tex1Dfetch(t_trisA, triAddr + 2);
                        float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
                        float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
                        float v = Oy + t*Dy;

                        if (v >= 0.0f && u + v <= 1.0f)
                        {
                            // Record intersection.
                            // Closest intersection not required => terminate.

                            hitT = t;
                            hitIndex = triAddr;
                            if (anyHit)
                            {
                                nodeAddr = EntrypointSentinel;
                                break;
                            }
                        }
                    }
                }
            } // triangle

            // Another leaf was postponed => process it as well.

            leafAddr = nodeAddr;
            if(nodeAddr<0)
            {
                nodeAddr = *(int*)stackPtr;
                stackPtr -= 4;
            }
        } // leaf
    } // traversal

    // Remap intersected triangle index, and store the result.

    if (hitIndex != -1)
        hitIndex = tex1Dfetch(t_triIndices, hitIndex);
    STORE_RESULT(rayidx, hitIndex, hitT);
}

//------------------------------------------------------------------------
